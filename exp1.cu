#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""  
  
#include <stdio.h>  
  
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);  
void printDevProp(hipDeviceProp_t &devProp);

__global__ void addKernel(int *c, const int *a, const int *b)  
{
    // we only give 1 dimension of size so that only x value matters.
    int i = threadIdx.x;  
    c[i] = a[i] + b[i];  
}  
  
int main()  
{  
    const int arraySize = 5;  
    const int a[arraySize] = { 1, 2, 3, 4, 5 };  
    const int b[arraySize] = { 10, 20, 30, 40, 50 };  
    int c[arraySize] = { 0 };  
 
    // Add vectors in parallel.  
    hipError_t cudaStatus;  
    int num = 0;  
    hipDeviceProp_t prop;  
    cudaStatus = hipGetDeviceCount(&num);  
    for(int i = 0;i<num;i++)  
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i); 
        hipGetDeviceProperties(&prop,i);
        printDevProp(prop);  
    }  
 
    // Add vectors in parallel.  
    cudaStatus = addWithCuda(c, a, b, arraySize);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "addWithCuda failed!");  
        return 1;  
    }  
  
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",  
        c[0], c[1], c[2], c[3], c[4]);  
  
    // hipDeviceReset must be called before exiting in order for profiling and  
    // tracing tools such as Nsight and Visual Profiler to show complete traces.  
    cudaStatus = hipDeviceReset();  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipDeviceReset failed!");  
        return 1;  
    }  
  
    return 0;  
}  
  
// Helper function for using CUDA to add vectors in parallel.  
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)  
{  
    int *dev_a = 0;  
    int *dev_b = 0;  
    int *dev_c = 0;  
    hipError_t cudaStatus;  
  
    // Choose which GPU to run on, change this on a multi-GPU system.  
    cudaStatus = hipSetDevice(0);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
        goto Error;  
    }  
  
    // Allocate GPU buffers for three vectors (two input, one output)    .  
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    // Copy input vectors from host memory to GPU buffers.  
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
    // Launch a kernel on the GPU with one thread for each element.  
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);  
  
    // hipDeviceSynchronize waits for the kernel to finish, and returns  
    // any errors encountered during the launch.  
    cudaStatus = hipDeviceSynchronize();  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);  
        goto Error;  
    }  
  
    // Copy output vector from GPU buffer to host memory.  
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
Error:  
    hipFree(dev_c);  
    hipFree(dev_a);  
    hipFree(dev_b);  
      
    return cudaStatus;  
}


// Print device properties
void printDevProp(hipDeviceProp_t &devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}
